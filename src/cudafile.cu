#include "hip/hip_runtime.h"
#include<thrust/host_vector.h>
#include<thrust/device_vector.h>
#include<thrust/device_ptr.h>

#define blocksize 512
#define maxblocks 65535

__global__ void funcKernel(int size, int* a1, int* a2, int* a3)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    while (i < size)
    {
        a3[i]=a1[i]+a2[i];
    }
}

void func(int size, int* a1, int* a2, int* a3)
{
    int gridsize = size / blocksize + 1;
    if (gridsize > maxblocks) gridsize = maxblocks;

    funcKernel << <gridsize, blocksize >> > (size, a1, a2, a3);
}

void FillWithValue(int* arr, int size, int val)
{

    thrust::device_ptr<int> d = thrust::device_pointer_cast(arr);
    thrust::fill(d, d + size, val);
}