#include "hip/hip_runtime.h"
#include<thrust/host_vector.h>
#include<thrust/device_vector.h>
#include<thrust/device_ptr.h>

#define blocksize 512
#define maxblocks 65535
#define N 1024

__global__ void addKernel(int* a, int* b, int* c){
    int i = blockIdx.x;

    if(i < N){
        c[i] = a[i] + b[i];
    }
}

void add(int* a, int* b, int* c){
    addKernel<<<N,1>>>( a, b, c );
}